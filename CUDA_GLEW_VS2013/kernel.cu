#include "hip/hip_runtime.h"
#define GLEW_STATIC
#include <GL/glew.h>
#include <GL/glut.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include "cuda_gl_interop.h"
#include "hip/hip_vector_types.h"
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <ctime>
#include <iostream>

int width = 800, height = 600;
float vboWindowScale = 0.5f;
int widthScaled, heightScaled;
double fovy = 60.0, aspect, zNear = 1.0, zFar = 1024.0;
unsigned int FPS = 60, msecs = 1000 / FPS;
double eyeX, eyeY, eyeZ;
bool animation = true;

GLfloat xRotated, yRotated, zRotated;
GLuint R, G, B = 0;

//ilosc kolorow
const int csize = 7;
//kolory dla obiektow
int4 colors[csize];

unsigned int vertVBO = 0, normalVBO = 0;
struct hipGraphicsResource *cudaVertVBO = NULL, *cudaNormalVBO = NULL;
int vertSize, normalSize;

unsigned int pbo = 0;
struct hipGraphicsResource *cudaPBO = NULL;
int pboSize;

void initialize();
void resetCamera();
void createVBO();
void deleteVBO();
void recreateVBO();
void createPBO();
void deletePBO();
void recreatePBO();
int exitHandler();
void display();
void reshape(int w, int h);
void keyboard(unsigned char key, int x, int y);
void special(int key, int x, int y);
void timer(int value);
void displayRGB();
void drawGlutObject(int id, GLdouble size, GLfloat X, GLfloat Y, GLfloat Z, int4 color);
void drawTeaPot(int size, GLfloat X, GLfloat Y, GLfloat Z);
void drawSphere(GLdouble size, GLfloat X, GLfloat Y, GLfloat Z);
void drawCube(GLdouble size, GLfloat X, GLfloat Y, GLfloat Z);
void drawTetrahedron(GLdouble size, GLfloat X, GLfloat Y, GLfloat Z);
void randCol(int4[], int size);
 
__device__ __forceinline__ int segmentation(int value, int prog)
{
	return (value < prog) ? 0 : value;
}


__global__ void fancyKernel(uchar3 *pixels,int width, int height, int R,  int G, int B)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;


	if ((x < width) && (y < height))
	{
		int i = y*width + x;

		
		pixels[i].x = segmentation(pixels[i].x, R); // R
		pixels[i].y = segmentation(pixels[i].y, G); // G
		pixels[i].z = segmentation(pixels[i].z, B); // B
		
	}
}

int main(int argc, char *argv[])
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE | GLUT_DEPTH);
	glutInitWindowSize(width, height);
	glutCreateWindow(argv[0]);
	initialize();
	glutDisplayFunc(display);
	glutReshapeFunc(reshape);
	glutKeyboardFunc(keyboard);
	glutSpecialFunc(special);
	glutTimerFunc(msecs, timer, 0);
	glutMainLoop();
	return 0;
}

void initialize()
{
	//kolory
	GLfloat mat_specular[] = { 1.0f, 1.0f, 1.0f, 1.0f };
	GLfloat mat_shininess[] = { 50.0f };
	GLfloat light_position[] = { 1.0f, 1.0f, 1.0f, 0.0f };
	randCol(colors, csize);

	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
	glEnable(GL_DEPTH_TEST);
	glEnable(GL_POINT_SMOOTH);
	glEnable(GL_LINE_SMOOTH);
	glPointSize(2.0f);
	glLineWidth(2.0f);


	glShadeModel(GL_SMOOTH);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);

	glColorMaterial(GL_FRONT_AND_BACK, GL_AMBIENT_AND_DIFFUSE);
	resetCamera();

	if (hipSetDevice(0) != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		exitHandler();
		exit(EXIT_FAILURE);
	}

	glewExperimental = true;
	if (glewInit() != GLEW_OK)
	{
		fprintf(stderr, "GLEW initialization failed!");
		exitHandler();
		exit(EXIT_FAILURE);
	}

	/*if (glewIsSupported("GL_VERSION_2_0") == false)
	{
	fprintf(stderr, "Extensions are not supported!");
	exitHandler();
	exit(EXIT_FAILURE);
	}*/

	//utworzenie bufor�w wierzcho�k�w
	createVBO();
	//utworzenie Pixel Buffer Object
	createPBO();
}

void resetCamera()
{
	eyeX = 2.0;
	eyeY = 2.0;
	eyeZ = 2.0;
}



void createVBO()
{
	widthScaled = int(width*vboWindowScale);
	heightScaled = int(height*vboWindowScale);



	vertSize = 4 * widthScaled*heightScaled;

	//utworzenie identyfikatora obiektu buforowego
	glGenBuffers(1, &vertVBO);
	//dowiazanie identyfikatora do obiektu buforowego
	//GL_ARRAY_BUFFER - obiekt buforowy tablic wierzcholkow
	glBindBuffer(GL_ARRAY_BUFFER, vertVBO);
	//ladowanie danych do obiektu buforowego
	//GL_DYNAMIC_DRAW - wielokrotne pobieranie danych i wielokrotne ich wykorzystanie do zapisu do obiektu OpenGL,
	glBufferData(GL_ARRAY_BUFFER, vertSize*sizeof(float), 0, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_ARRAY_BUFFER, 0);

	//rejestacja bufora OpenGL
	hipGraphicsGLRegisterBuffer(&cudaVertVBO, vertVBO, hipGraphicsRegisterFlagsNone);


	normalSize = 3 * widthScaled*heightScaled;
	glGenBuffers(1, &normalVBO);
	glBindBuffer(GL_ARRAY_BUFFER, normalVBO);
	glBufferData(GL_ARRAY_BUFFER, normalSize*sizeof(float), 0, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	hipGraphicsGLRegisterBuffer(&cudaNormalVBO, normalVBO, cudaGraphicsMapFlagsWriteDiscard);
}

//ususuwanie obiektow VBO
void deleteVBO()
{
	hipGraphicsUnregisterResource(cudaVertVBO);
	cudaVertVBO = NULL;
	glDeleteBuffers(1, &vertVBO);
	vertVBO = 0;

	hipGraphicsUnregisterResource(cudaNormalVBO);
	cudaNormalVBO = NULL;
	glDeleteBuffers(1, &normalVBO);
	normalVBO = 0;
}


void recreateVBO()
{
	deleteVBO();
	createVBO();
}


// Utworzenie Pixel Buffer Object
// Rodzaj buforu OpenGl, s�uzacy do przechowywania pikseli
void createPBO()
{
	pboSize = 3 * width*height;
	glGenBuffers(1, &pbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
	glBufferData(GL_PIXEL_UNPACK_BUFFER, pboSize*sizeof(char), 0, GL_DYNAMIC_COPY);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
	hipGraphicsGLRegisterBuffer(&cudaPBO, pbo, hipGraphicsRegisterFlagsNone);
}
void deletePBO()
{
	hipGraphicsUnregisterResource(cudaPBO);
	cudaPBO = NULL;
	glDeleteBuffers(1, &pbo);
	pbo = 0;
}
void recreatePBO()
{
	deletePBO();
	createPBO();
}

int exitHandler()
{
	deleteVBO();
	deletePBO();
	if (hipDeviceReset() != hipSuccess)
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return EXIT_FAILURE;
	}

	return EXIT_SUCCESS;
}

void display()
{
	hipError_t err = hipSuccess;
	uchar3 *pixels = NULL;

	size_t num_bytes;
	dim3 block_dim(16, 16);
	dim3 grid_dim((width + block_dim.x - 1) / block_dim.x, (height + block_dim.y - 1) / block_dim.y);
	dim3 grid_dim_scaled((widthScaled + block_dim.x - 1) / block_dim.x, (heightScaled + block_dim.y - 1) / block_dim.y);

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glLoadIdentity();

	//define a viewing transformation
	gluLookAt(eyeX, eyeY, eyeZ,
		0.0, 0.0, 0.0,
		0.0, 1.0, 0.0);

	glMatrixMode(GL_MODELVIEW);
	// czysczenie bufora rysowania
	glClear(GL_COLOR_BUFFER_BIT);
	////
	glLoadIdentity();

	
	drawGlutObject(1, 0.5, 0.0, 0.0, -3.5, colors[0]);
	drawGlutObject(2, 0.5, 1.0, 0.0, -3.5, colors[1]);
	drawGlutObject(3, 0.5, -1.0, 1.0, -2.5, colors[2]);
	drawGlutObject(4, 0.5, 1.0, 1.0, -4.5, colors[3]);
	drawGlutObject(5, 0.5, -1.0, -1.0, -4.5, colors[4]);
	drawGlutObject(6, 0.5, -1.0, 2.0, -3.5, colors[5]);
	drawGlutObject(7, 0.5, 0.5, 1.0, -7.5, colors[6]);
	drawGlutObject(8, -0.5, 0.5, -1.0, -7.5, colors[6]);
	// PBO.
	glBindBuffer(GL_PIXEL_PACK_BUFFER, pbo);
	glReadPixels(0, 0, width, height, GL_RGB, GL_UNSIGNED_BYTE, 0);
	glBindBuffer(GL_PIXEL_PACK_BUFFER, 0);

	hipGraphicsMapResources(1, &cudaPBO, 0);
	hipGraphicsResourceGetMappedPointer((void**)&pixels, &num_bytes, cudaPBO);

	fancyKernel << <grid_dim, block_dim >> >(pixels, width, height, R, G, B);
	err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "fancyKernel kernel launch failed: %s\n", hipGetErrorString(err));
		exitHandler();
		exit(EXIT_FAILURE);
	}
	err = hipDeviceSynchronize();
	if (hipSuccess != err)
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching fancyKernel kernel!\n", err);
		exitHandler();
		exit(EXIT_FAILURE);
	}

	hipGraphicsUnmapResources(1, &cudaPBO, 0);

	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
	glDrawPixels(width, height, GL_RGB, GL_UNSIGNED_BYTE, 0);

	

	glutSwapBuffers();
}

void reshape(int w, int h)
{
	width = (w > 0) ? w : 1;
	height = (h > 0) ? h : 1;
	aspect = (double)width / (double)height;

	glViewport(0, 0, width, height);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluPerspective(fovy, aspect, zNear, zFar);
	glMatrixMode(GL_MODELVIEW);
	recreateVBO();
	recreatePBO();
}

void keyboard(unsigned char key, int x, int y)
{
	switch (key)
	{
	case  'q': (R < 255) ? R += 1 : 255; break;
	case  'a': (R > 0) ? R -= 1 : 0; break;
	case  'w': (G < 255) ? G += 1 : 255; break;
	case  's': (G > 0) ? G -= 1 : 0;  break;
	case  'e': (B < 255) ? B += 1 : 255; break;
	case  'd': (B > 0) ? B -= 1 : 0;  break;
	case 'r':
	case 'R': resetCamera(); break;
	case 32:
	{
			   if (animation = !animation)
			   {
				   glutTimerFunc(msecs, timer, 0);
			   }
			   break;
	}
	case 27: exit(exitHandler()); break;
	default:;
	}
	glutPostRedisplay();

	/*wyswietlanie wartosci RGB*/
	system("cls");
	std::cout << "R: " << R << std::endl << "G: " << G << std::endl << "B: " << B << std::endl;

}

void special(int key, int x, int y)
{
	switch (key)
	{
	case GLUT_KEY_LEFT: eyeX -= 0.5; break;
	case GLUT_KEY_RIGHT: eyeX += 0.5; break;
	case GLUT_KEY_UP: eyeY += 0.5; break;
	case GLUT_KEY_DOWN: eyeY -= 0.5; break;
	case GLUT_KEY_HOME: eyeZ -= 0.5; break;
	case GLUT_KEY_END: eyeZ += 0.5; break;
	default:;
	}


	glutPostRedisplay();
}

void timer(int value)
{
	if (animation)
	{
		glutPostRedisplay();
		glutTimerFunc(msecs, timer, 0);
	}
}


void randCol(int4 color[], int size) {
	srand(time(NULL));
	for (int i = 0; i < size; i++) {
		int temp = rand() % 255;
		color[i] = { rand() % 255, rand() % 255, rand() % 255, rand() % 255 };
	}
}

void drawTeaPot(int size, GLfloat X, GLfloat Y, GLfloat Z) {
	glPushMatrix();
	glTranslatef(X, Y, Z);
	glRotatef(90, 0.1, 0.2, 0.5);
	glColor3ub(0, 255, 0);
	glutSolidTeapot(size);
	glPopMatrix();
}


void drawGlutObject(int id, GLdouble size, GLfloat X, GLfloat Y, GLfloat Z, int4 color) {
	glPushMatrix();
	glTranslatef(X, Y, Z);
	glRotatef(90, 0.1, 0.2, 0.5);
	glColor3ub(color.x, color.y, color.z);

	switch (id)
	{
	case 1: glutSolidTeapot(size); break;
	case 2: glutSolidSphere(size, 50, 50); break;
	case 3: glutSolidCube(size); break;
	case 4: glutSolidTetrahedron(); break;
	case 5: glutSolidIcosahedron(); break;
	case 6: glutSolidOctahedron(); break;
	case 7: glutSolidDodecahedron(); break;
	case 8:	glutSolidTorus(size, 10, 1, 1);
	default:;
	}

	glPopMatrix();


}



void drawSphere(GLdouble size, GLfloat X, GLfloat Y, GLfloat Z) {
	glPushMatrix();
	glTranslatef(X, Y, Z);
	glRotatef(90, 0.1, 0.2, 0.5);
	glColor3ub(0, 255, 0);
	glutSolidSphere(size, 50, 50);
	glPopMatrix();
}

void drawCube(GLdouble size, GLfloat X, GLfloat Y, GLfloat Z) {
	glPushMatrix();//
	glTranslatef(X, Y, Z);
	glRotatef(90, 0.1, 0.2, 0.5);
	glColor3ub(0, 255, 0);
	glutSolidCube(size);
	glPopMatrix();
}

void drawTetrahedron(GLdouble size, GLfloat X, GLfloat Y, GLfloat Z) {
	glPushMatrix();
	glTranslatef(X, Y, Z);
	glRotatef(90, 0.1, 0.2, 0.5);
	glColor3ub(0, 255, 0);
	glutSolidTetrahedron();
	glPopMatrix();
}